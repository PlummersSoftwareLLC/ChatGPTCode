// (Davepl) Write a 20-line explanation of how this program works for the next programmer who has to look at it.  Write it as a C++ comment.

/* 
    This program is a segmented prime sieve that makes use of the CUDA API to perform the sieve calculation on the GPU. 
    The program first allocates the necessary memory on the CPU and then transfers the memory to the GPU. 
    The primeSieveKernel function, which is executed on the GPU, performs the sieve calculation by marking multiples 
    of prime numbers as non-prime (1). Stream compaction is used to compact the list of prime numbers into an array
    of only prime numbers, which is then copied back to the CPU for processing. The for loop in the primeSieveKernel 
    function iterates from 3 to sqrt(n) and marks multiples of prime numbers as non-prime. The segSieveKernel function 
    is used to divide the sieve calculation into segments and evenly distribute the work among multiple CUDA threads. 
    The final result of the program is a list of all prime numbers up to a specified number (n).
*/

// (Davepl) Now rewrite it with the comment symbol in front of each line, and use a surfer-dude dialect

// Yo dude, this here code is a prime sieve written in C++, but it's super chill cuz it uses CUDA API to run on the GPU.
// The main idea is that we wanna find all the prime numbers up to some big number, say 100 million.
// But like, finding all those primes can be a real drag on the CPU.
// So, we hand off the heavy lifting to the GPU, which is way faster, man.
// The trick is to break up the 100 million into smaller chunks, or "segments", and then run the sieve on each segment.
// Then we take all those prime numbers we found and use them to sieve out the next segment, and so on.
// The end result is that we have all the primes up to 100 million, and we did it super fast, dude.
//
// Alright, let's "dive" in.
// First up, we got the main function. It sets everything up, including allocating memory on the GPU.
// Then we got the primeSieveKernel function, which is what actually does the sieve on the GPU.
// This function takes in the start and end of the segment we're working on, and a pointer to the memory we allocated on the GPU.
// Inside the kernel, we use a for loop to check each number in the segment and see if it's prime.
// If it's not prime, we set it to 0 in the GPU memory.
// Once the kernel is done, we use cudaMemcpy to copy the GPU memory back to the CPU so we can see the results.
//
// Finally, we wrap it up with a unified memory version, which is even more dope cuz it
// ..
// [network error]
// ..
// automatically handles the memory transfer between CPU and GPU.
// Basically, we just declare the variables we need as "unified memory" and the GPU and CPU can access it as needed.
// No need to use cudaMemcpy, it's all handled behind the scenes, bro.
// That's it, man. Just a quick rundown of this rad prime sieve program that's blazing fast on the GPU.  Broh.

#include <iostream>
#include <cmath>
#include <hip/hip_runtime.h>

const int n = 100000000;                

#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    std::cerr << "Error: " << hipGetErrorString(x) << std::endl; \
    exit(EXIT_FAILURE);}} while(0)

__global__ void primeSieveKernel(int *numbers, int sqrt_n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < sqrt_n) {
        int j = 2 * i + 3;
        for (int k = i + j; k < (n / 2); k += j) {
            numbers[k] = 1;
        }
    }
}

int main(int argc, char *argv[]) {
    
    int sqrt_n = (int)sqrt(n);
    int blockSize = 256;

    // Allocate unified memory
    int *_numbers;
    CUDA_CALL(hipMallocManaged(&_numbers, (n / 2) * sizeof(int)));

    // Set initial values for numbers
    for (int i = 0; i < (n / 2); i++)
        _numbers[i] = 0;

    dim3 block(blockSize, 1, 1);
    dim3 grid((int)ceil(sqrt_n / (double)blockSize), 1, 1);

    // Launch the kernel on the GPU
    primeSieveKernel<<<grid, block>>>(_numbers, sqrt_n);

    // Wait for the GPU to finish
    CUDA_CALL(hipDeviceSynchronize());

    // Print the number of primes
    int count = 1;
    for (int i = 0; i < n / 2; i++)
        count += (_numbers[i] == 0);

    std::cout << count << " primes found." << std::endl;

    // Free the memory
    CUDA_CALL(hipFree(_numbers));
    return 0;
}